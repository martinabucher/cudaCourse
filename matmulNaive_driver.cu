#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#include "helper.cuh"

#define DIM_TILE   32
#define DIM_GRID   64
#define DIM    (DIM_TILE*DIM_GRID)

__global__ void matmul_naive(int N, const float *A, const float *B, float *AB);
__global__ void matmul_tiled(int N, const float *A, const float *B, float *AB);

int main(){

  int best_device=get_best_device();
  errCheck(hipSetDevice(best_device));
  
  dim3 gridDim (DIM_GRID, DIM_GRID,   1);
  dim3 blockDim(DIM_TILE, DIM_TILE,  1);

  printf("Block dim = ( %d , %d )\n", blockDim.x, blockDim.y );

  float *a_h, *b_h, *ab_h;

  a_h =(float*) malloc(DIM*DIM*sizeof(float)); 
  b_h =(float*) malloc(DIM*DIM*sizeof(float)); 
  ab_h=(float*) malloc(DIM*DIM*sizeof(float)); 

  if ( a_h == NULL || b_h == NULL || ab_h == NULL ){
    fprintf(stderr,"Error: malloc failed. Exiting.\n"); 
    exit(-1);  //  'exit' is defined in stdlib.h --- the program is terminated with return status 0 here, meaning successful completion. 
  }

  // Now we use the random number generator of the C Standard Library to generte random input data. 

  srand(676);    //

  for(int i=0; i<DIM*DIM; i++){
    a_h[i]=((float) rand())/((float) RAND_MAX);
    b_h[i]=((float) rand())/((float) RAND_MAX);
  } 

 // The rand() random number generator produces a sequence of integer pseudo-random numbers from 0 to RAND_MAX (inclusive). 
 // This is not guaranteed to be a good random number generator and in many implementations is not. The seed is set by
 // by the call void srand( unsigned seed ), so each time this program is run the same sequence of pseudo-random numbers will result.
 // Seee https://en.cppreference.com/w/c/numeric/random/rand

  float *a_d, *b_d, *ab_d;

  errCheck(hipMalloc((void**) &a_d, DIM*DIM*sizeof(float)));
  errCheck(hipMalloc((void**) &b_d, DIM*DIM*sizeof(float)));
  errCheck(hipMalloc((void**) &ab_d,DIM*DIM*sizeof(float)));

  errCheck(hipMemcpy(a_d, a_h, DIM*DIM*sizeof(float), hipMemcpyHostToDevice));
  errCheck(hipMemcpy(b_d, b_h, DIM*DIM*sizeof(float), hipMemcpyHostToDevice));
  
  float time_ms; hipEvent_t startBis, stopBis;
  errCheck(hipEventCreate(&startBis));
  errCheck(hipEventCreate(&stopBis));

  errCheck(hipEventRecord(startBis, 0));
  matmul_naive<<<gridDim,blockDim>>>(DIM, a_d, b_d, ab_d); 
  hipError_t err = hipGetLastError(); // ERROR CHECKING
  errCheck(hipEventRecord(stopBis, 0));
  if ( err != hipSuccess ){
     fprintf(stderr,"CUDA Error: %s\n", hipGetErrorString(err));   
     exit(-1); 
  }  
  errCheck(hipEventSynchronize(stopBis)); 

  errCheck(hipEventElapsedTime(&time_ms, startBis, stopBis));
  printf("Device timing (in milliseconds) = %g\n", time_ms);

  errCheck(hipDeviceSynchronize());
  errCheck(hipMemcpy(ab_h, ab_d, DIM*DIM*sizeof(float), hipMemcpyDeviceToHost));
     // same syntax as above and last argument indicates direction of transfer 

  // Compare result with host function 

  clock_t start=clock();
  float* ab_h2=(float*) malloc(DIM*DIM*sizeof(float));
  for(int i=0; i<DIM; i++)
    for(int j=0; j<DIM; j++){
      float sum=0.0;
      for(int k=0; k<DIM; k++)
         sum+=a_h[i*DIM+k]*b_h[k*DIM+j]; 
      ab_h2[i*DIM+j]=sum;
    }
  clock_t end=clock();
  float timing= ( (float) (end-start) )/( (float) CLOCKS_PER_SEC );
  timing/=1.e-3; 
  printf("Host function time= %e millisecs. \n",timing);
  printf("CLOCKS_PER_SEC= %e\n", (float) CLOCKS_PER_SEC);
  float speedup_factor=timing/time_ms;
  printf("Speedup factor = %e\n", speedup_factor);

  // Compare results
  float eps=1.e-3;
  bool same=true;
  for(int i=0; i<DIM; i++)
    for(int j=0; j<DIM; j++){
      if ( fabs( ab_h[i*DIM +j] - ab_h2[i*DIM+j] ) > eps )
        same = false; 
    }
  if (same == true) 
    printf("The host and device calculations agree.\n");
  else 
    printf("The host and device calculations do not agree.\n"); 
  return 0; 
} 


__global__ void matmul_naive(int N, const float *A, const float *B, float *AB) {
  // compute position in AB that this thread is responsible for
  const uint x = blockIdx.x * blockDim.x + threadIdx.x;
  const uint y = blockIdx.y * blockDim.y + threadIdx.y;
  // `if` condition is necessary for N is not a multiples of 32
  if (x < N && y < N) {
    float sum = 0.0;
    for (int i = 0; i < N; ++i)
      sum += A[x * N + i] * B[i * N + y];
    AB[x * N + y] = sum;
  }
}


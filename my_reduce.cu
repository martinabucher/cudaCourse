

#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ void reduce_sum(float *s, const int n, float *sum);
__global__ void testReduceSum(float* s, const int n, float *result);

__device__ void reduce_sum(float *s, const int n, float *sum){
  if (n == 1){   // eliminate trivial case 
     *sum = s[0]; 
  }
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  int step=1;
  while(1){
    if (2*step >= n ) break;
    step*=2;
  }
  if (index == 0 )
     printf("step %d\n",step); 
  int n_max=n; 
  while(1){
    if ( index + step < n_max )
      s[index]+=s[index+step]; 
    __syncthreads();
    n_max=step;
    if ( step==1 ) break;
    step/=2;
  }
  *sum=s[0];
}

__global__ void testReduceSum(float* s, const int n, float *result){
 reduce_sum(s, n, result);
}

int main(void){
  const int thread_dim=10;
  const int block_dim =10;
  int n=100; 
  size_t size=n*sizeof(float);
  float* s_d; 
  float *result_d;
  float result_h;
  hipMalloc((void **) &s_d, size);
  hipMalloc((void **) &result_d,sizeof(float));
  float s_h[size];
  float local_sum=0.; 
  for(int j=0;j<n;j++){
    s_h[j]=(float) (j+1); 
    local_sum+=s_h[j];}
  printf("Expected sum is equal to %f \n", local_sum);
  hipMemcpy(s_d, s_h, size, hipMemcpyHostToDevice);
  testReduceSum<<<block_dim,thread_dim>>>(s_d,n,result_d); 
  hipMemcpy(&result_h, result_d, sizeof(float), hipMemcpyDeviceToHost);
  printf("CUDA sum is equal to %f \n", result_h);
}

